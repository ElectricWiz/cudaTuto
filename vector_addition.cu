
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

__global__ void device_add(int* a, int* b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void host_add(int* a, int* b, int* c) {
    for(int idx=0;idx<N;idx++) {
        c[idx] = a[idx] + b[idx];
    }
}

void fill_array(int* data) {
    for(int idx=0;idx<N;idx++) {
        data[idx] = idx;
    }
}

void print_output(int* a, int* b, int* c) {
    for(int idx=0;idx<N;idx++) {
        printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
    }
}

int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    int threads_per_block = 8;
    int no_of_blocks = N/threads_per_block;

    a = (int *)malloc(size); fill_array(a);
    b = (int *)malloc(size); fill_array(b);
    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, N*sizeof(int));
    hipMalloc((void **)&d_b, N*sizeof(int));
    hipMalloc((void **)&d_c, N*sizeof(int));

    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    device_add<<<no_of_blocks,threads_per_block>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    print_output(a,b,c);

    free(a); free(b); free(c);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}