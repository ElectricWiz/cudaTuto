
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
    printf("Hello from GPU! from thread {%d,%d} From device %d\n", threadIdx.x,blockIdx.x);
}

int main(void) {
    printf("Hello world from Host!\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}